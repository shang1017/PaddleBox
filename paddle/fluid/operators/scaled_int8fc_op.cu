#include "hip/hip_runtime.h"
/* Copyright (c) 2020 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <string>
#include "paddle/fluid/framework/eigen.h"
#include "paddle/fluid/operators/scaled_int8fc_op.h"
#include "paddle/phi/kernels/funcs/blas/blas.h"
#include "paddle/fluid/platform/device/gpu/gpu_primitives.h"
#include "paddle/fluid/platform/device/gpu/gpu_info.h"

using GPUCtx = phi::GPUContext;
namespace paddle {
namespace operators {
using framework::Tensor;

#define CUDA_KERNEL_LOOP(i, n)                                 \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); \
       i += blockDim.x * gridDim.x)

const int CUDA_NUM_THREADS = paddle::platform::PADDLE_CUDA_NUM_THREADS;
static inline int GET_BLOCKS(const int N) {
  return (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS;
}

// cast to fp16 & padding
template <typename T>
__global__ void kernel_cast_and_padding(const int N, 
                      const unsigned int rown_ori, const unsigned int coln_ori,
                      const unsigned int rown_pad, const unsigned int coln_pad,
                      const T* matrix, int8_t* matrix_pad,
                      T grad_scale_factor,
                      T expand_factor,
                      T clip,
                      float int8_range) {
  CUDA_KERNEL_LOOP(i, N) { 
      int col_idx = i % coln_pad;
      int row_idx = i / coln_pad;
      if (row_idx < rown_ori && col_idx < coln_ori) {
          int idx = row_idx * coln_ori + col_idx;
          //matrix_pad[i] = static_cast<paddle::platform::float16>(matrix[idx] * grad_scale_factor);

          // mul expand_factor, mul10.0
          T matrix_idx_expand = matrix[idx] *  expand_factor;
          // range[-2.0, 2.0]
          T matrix_idx_expand_clip = 0.0;
          if ((matrix_idx_expand - 0) >= 1e-6) {
               matrix_idx_expand_clip = (matrix_idx_expand - clip) > 1e-6 ? clip : matrix_idx_expand;
          } else {
               matrix_idx_expand_clip = (matrix_idx_expand - (-clip)) < 1e-6 ? -clip : matrix_idx_expand;
          }
          // quantitate to int8
          T interval = (2*clip) / int8_range;
          matrix_pad[i] = static_cast<int8_t>(matrix_idx_expand_clip / interval + 0.50);
          //matrix_pad[i] = static_cast<int8_t>(matrix_idx_expand_clip / interval);
          //if (i % 1000000 == 0) {
          //  printf("rown_ori=%d, coln_ori=%d, rown_pad=%d, coln_pad=%d, i=%d, idx=%d,  matrix[idx]=%f, matrix_idx_expand=%f, matrix_idx_expand_clip=%f, matrix_pad_i=%d, end.\n", rown_ori, coln_ori, rown_pad, coln_pad, i, idx, matrix[idx], matrix_idx_expand, matrix_idx_expand_clip, static_cast<int>((matrix_pad[i])));
          //}
      } else {
          //matrix_pad[i] = static_cast<paddle::platform::float16>(0.0);
          matrix_pad[i] = static_cast<int8_t>(0);
      }
  }
}

template <typename T>
void cast_and_padding(hipStream_t stream, 
                      const unsigned int rown_ori, const unsigned int coln_ori,
                      const unsigned int rown_pad, const unsigned int coln_pad, 
                      const T* matrix, int8_t* matrix_pad,
                      T grad_scale_factor,
                      T expand_factor,
                      T clip,
                      float int8_range) {
                      //T grad_scale_factor = static_cast<T>(1.0)) {
  int N = rown_pad * coln_pad;
  kernel_cast_and_padding<<<GET_BLOCKS(N), CUDA_NUM_THREADS, 0, stream>>>(
      N, rown_ori, coln_ori, rown_pad, coln_pad, matrix, matrix_pad, grad_scale_factor, expand_factor, clip, int8_range);
}
// end cast to fp16 & padding


// cast to fp32 & cut
template <typename T>
__global__ void kernel_cast_and_cut(const int N, 
                      const unsigned int rown_ori, const unsigned int coln_ori,
                      const unsigned int rown_pad, const unsigned int coln_pad,
                      T* matrix, float* matrix_pad,
                      T scale_factor,
                      T expand_factor,
                      T interval) {
  CUDA_KERNEL_LOOP(i, N) { 
      int col_idx = i % coln_ori;
      int row_idx = i / coln_ori;
      int idx = row_idx * coln_pad + col_idx;

      matrix[i] = static_cast<T>(matrix_pad[idx]);
      matrix[i] /= expand_factor;
      matrix[i] *= interval;
      //if( i % 1000000 == 0) {
      //  printf("rown_ori=%d, coln_ori=%d, rown_pad=%d, coln_pad=%d, i=%d, matrix[i]=%f, idx=%d, fcget_matrix_pad_idx=%f, expand_factor=%f, interval=%f, end.\n", rown_ori, coln_ori, rown_pad, coln_pad, i, matrix[i], idx, matrix_pad[idx], expand_factor, interval);
    //}
  }
}

//cast_and_cut<T>(ctx.cuda_device_context().stream(), insnum_ori, outfea_ori, insnum_pad, outfea_pad, output->data<T>(), output_help.data<int>(), scale_factor, expand_factor, interval);
//cast_and_cut<T>(ctx.cuda_device_context().stream(), insnum_ori, outfea_ori, insnum_pad, outfea_pad, output->data<T>(), output_help.data<float>(), scale_factor, expand_factor, interval);
template <typename T>
void cast_and_cut(hipStream_t stream, 
                      const unsigned int rown_ori, const unsigned int coln_ori,
                      const unsigned int rown_pad, const unsigned int coln_pad, 
                      T* matrix, float* matrix_pad,
                      T scale_factor,
                      T expand_factor,
                      T weight_expand_factor,
                      T interval) {
  int N = rown_ori * coln_ori;
  //T expand_factor_double = expand_factor * expand_factor;
  T expand_factor_double = expand_factor * weight_expand_factor;
  kernel_cast_and_cut<<<GET_BLOCKS(N), CUDA_NUM_THREADS, 0, stream>>>(
      N, rown_ori, coln_ori, rown_pad, coln_pad, matrix, matrix_pad, scale_factor, expand_factor_double, interval);
}
// end cast to fp32 & cut

// add the same row vector to all matrix rows
template <typename T>
__global__ void kernel_vec_mat_row_add(const int N, const unsigned int rown,
                                       const unsigned int coln, T* matrix,
                                       const T* vector, const T bias_scale_factor_use) {
  //CUDA_KERNEL_LOOP(i, N) { matrix[i] += vector[i % coln] * bias_scale_factor_use; }
  CUDA_KERNEL_LOOP(i, N) { matrix[i] += vector[i % coln]; }
}

template <typename T>
void vec_mat_row_add(hipStream_t stream, const unsigned int rown,
                     const unsigned int coln, T* matrix, const T* vector, const T bias_scale_factor_use) {
  int N = rown * coln;
  kernel_vec_mat_row_add<<<GET_BLOCKS(N), CUDA_NUM_THREADS, 0, stream>>>(
      N, rown, coln, matrix, vector, bias_scale_factor_use);
}

// calculate col sum of a mat
template <typename T>
__global__ void kernel_add_col_sum_mat(const unsigned int rown,
                                       const unsigned int coln, const T* matrix,
                                       T* vector, const T bias_scale_factor_use) {
  CUDA_KERNEL_LOOP(i, coln) {
    for (unsigned int j = 0; j < rown; j++) {
      ////vector[i] += matrix[i * rown + j];
      //vector[i] += matrix[j * coln + i] * bias_scale_factor_use;
      //printf("i=%d, j=%d, rown=%d, coln=%d, matrix[j*coln+i]=%f, vector[i]=%f\n", i, j, rown, coln, matrix[j*coln+i], vector[i]);
      vector[i] += matrix[j * coln + i];
    }
  }
}

//col_sum_mat(stream, ins_num, dout_coln, dout->data<T>(), db->data<T>(), bias_scale_factor_use);
template <typename T>
void col_sum_mat(hipStream_t stream, const unsigned int rown,
                 const unsigned int coln, const T* matrix, T* vector,
                 const T bias_scale_factor_use) {
  kernel_add_col_sum_mat<<<GET_BLOCKS(coln), CUDA_NUM_THREADS, 0, stream>>>(
      rown, coln, matrix, vector, bias_scale_factor_use);
}
////// for grad /////
// cast to fp16 & padding
template <typename T>
__global__ void kernel_cast_and_padding_grad(const int N, 
                      const unsigned int rown_ori, const unsigned int coln_ori,
                      const unsigned int rown_pad, const unsigned int coln_pad,
                      const T* matrix, paddle::platform::float16* matrix_pad,
                      T grad_scale_factor) {
  CUDA_KERNEL_LOOP(i, N) { 
      int col_idx = i % coln_pad;
      int row_idx = i / coln_pad;
      if (row_idx < rown_ori && col_idx < coln_ori) {
          int idx = row_idx * coln_ori + col_idx;
          //matrix_pad[i] = static_cast<paddle::platform::float16>(matrix[idx]);
          matrix_pad[i] = static_cast<paddle::platform::float16>(matrix[idx] * grad_scale_factor);
      } else {
          matrix_pad[i] = static_cast<paddle::platform::float16>(0.0);
      }
  }
}

template <typename T>
void cast_and_padding_grad(hipStream_t stream, 
                      const unsigned int rown_ori, const unsigned int coln_ori,
                      const unsigned int rown_pad, const unsigned int coln_pad, 
                      const T* matrix, paddle::platform::float16* matrix_pad,
                      T grad_scale_factor) {
                      //T grad_scale_factor = static_cast<T>(1.0)) {
  int N = rown_pad * coln_pad;
  kernel_cast_and_padding_grad<<<GET_BLOCKS(N), CUDA_NUM_THREADS, 0, stream>>>(
      N, rown_ori, coln_ori, rown_pad, coln_pad, matrix, matrix_pad, grad_scale_factor);
}
// end cast to fp16 & padding


// cast to fp32 & cut
template <typename T>
__global__ void kernel_cast_and_cut_grad(const int N, 
                      const unsigned int rown_ori, const unsigned int coln_ori,
                      const unsigned int rown_pad, const unsigned int coln_pad,
                      T* matrix, paddle::platform::float16* matrix_pad,
                      T scale_factor) {
  CUDA_KERNEL_LOOP(i, N) { 
      int col_idx = i % coln_ori;
      int row_idx = i / coln_ori;
      int idx = row_idx * coln_pad + col_idx;
      matrix[i] = static_cast<T>(matrix_pad[idx]);
      matrix[i] *= scale_factor;
  }
}

template <typename T>
void cast_and_cut_grad(hipStream_t stream, 
                      const unsigned int rown_ori, const unsigned int coln_ori,
                      const unsigned int rown_pad, const unsigned int coln_pad, 
                      T* matrix, paddle::platform::float16* matrix_pad,
                      T scale_factor) {
  int N = rown_ori * coln_ori;
  kernel_cast_and_cut_grad<<<GET_BLOCKS(N), CUDA_NUM_THREADS, 0, stream>>>(
      N, rown_ori, coln_ori, rown_pad, coln_pad, matrix, matrix_pad, scale_factor);
}
// end cast to fp32 & cut

// add the same row vector to all matrix rows
template <typename T>
__global__ void kernel_vec_mat_row_add_grad(const int N, const unsigned int rown,
                                       const unsigned int coln, T* matrix,
                                       const T* vector, const T bias_scale_factor_use) {
  CUDA_KERNEL_LOOP(i, N) { matrix[i] += vector[i % coln] * bias_scale_factor_use; }
}

template <typename T>
void vec_mat_row_add_grad(hipStream_t stream, const unsigned int rown,
                     const unsigned int coln, T* matrix, const T* vector, const T bias_scale_factor_use) {
  int N = rown * coln;
  kernel_vec_mat_row_add_grad<<<GET_BLOCKS(N), CUDA_NUM_THREADS, 0, stream>>>(
      N, rown, coln, matrix, vector, bias_scale_factor_use);
}

// calculate col sum of a mat
template <typename T>
__global__ void kernel_add_col_sum_mat_grad(const unsigned int rown,
                                       const unsigned int coln, const T* matrix,
                                       T* vector, const T bias_scale_factor_use) {
  CUDA_KERNEL_LOOP(i, coln) {
    for (unsigned int j = 0; j < rown; j++) {
      ////vector[i] += matrix[i * rown + j];
      //vector[i] += matrix[j * coln + i] * bias_scale_factor_use;
      vector[i] += matrix[j * coln + i];
    }
  }
}

//col_sum_mat(stream, ins_num, dout_coln, dout->data<T>(), db->data<T>(), bias_scale_factor_use);
template <typename T>
void col_sum_mat_grad(hipStream_t stream, const unsigned int rown,
                 const unsigned int coln, const T* matrix, T* vector,
                 const T bias_scale_factor_use) {
  kernel_add_col_sum_mat_grad<<<GET_BLOCKS(coln), CUDA_NUM_THREADS, 0, stream>>>(
      rown, coln, matrix, vector, bias_scale_factor_use);
}

template <typename DeviceContext, typename T>
class ScaledINT8FCCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    //VLOG(0) << "begin compute.";
    auto* input = ctx.Input<framework::LoDTensor>("Input"); // framework::Tensor*
    auto* w = ctx.Input<Tensor>("W");
    auto* bias = ctx.Input<Tensor>("Bias");
    auto* output = ctx.Output<framework::LoDTensor>("Out");
    auto input_scale_factor = ctx.Attr<float>("input_scale_factor");
    auto bias_scale_factor = ctx.Attr<float>("bias_scale_factor");
    
    auto clip_factor = ctx.Attr<float>("clip_factor");
    auto expand_factor = ctx.Attr<float>("expand_factor");

    auto weight_clip_factor = ctx.Attr<float>("weight_clip_factor");
    auto weight_expand_factor = ctx.Attr<float>("weight_expand_factor");
    
    auto int8_range = ctx.Attr<float>("int8_range");

    auto input_dims = input->dims();
    auto w_dims = w->dims();
    auto ins_num = input_dims[0];  // oriinput: ins_num*in_feat, oriweight: in_feat* out_fea, output: ins_num* out_feat
    auto in_feat = input_dims[1];
    auto out_feat = w_dims[1];

    // get data ptr
    const T* in_data = input->data<T>();
    const T* w_data = w->data<T>();
    const T* bias_data = bias->data<T>();

    output->mutable_data<T>(ctx.GetPlace());
    output->Resize({ins_num, w_dims[1]});

    auto& dev_ctx = ctx.template device_context<GPUCtx>();
    // cast and pad
    const unsigned int insnum_ori = ins_num;
    const unsigned int infea_ori = in_feat;
    const unsigned int outfea_ori = out_feat;
    
    const unsigned int insnum_pad = (insnum_ori % 8) == 0 ? insnum_ori : insnum_ori + (8 - insnum_ori % 8);
    const unsigned int infea_pad = (infea_ori % 8) == 0 ? infea_ori : infea_ori + (8 - infea_ori % 8);
    const unsigned int outfea_pad = (outfea_ori % 8) == 0 ? outfea_ori : outfea_ori + (8 - outfea_ori % 8);

    framework::Tensor input_help;
    input_help = ctx.AllocateTmpTensor<int8_t, DeviceContext>({insnum_pad, infea_pad}, dev_ctx);

    framework::Tensor w_help;
    w_help = ctx.AllocateTmpTensor<int8_t, DeviceContext>({infea_pad, outfea_pad}, dev_ctx);

    framework::Tensor bias_help;
    bias_help = ctx.AllocateTmpTensor<int8_t, DeviceContext>({outfea_pad, 1}, dev_ctx);

    framework::Tensor output_help;
    output_help = ctx.AllocateTmpTensor<float, DeviceContext>({insnum_pad, outfea_pad}, dev_ctx);

    T scale = static_cast<T>(1.0);
    expand_factor = static_cast<T>(expand_factor);
    clip_factor = static_cast<T>(clip_factor);

    weight_expand_factor = static_cast<T>(weight_expand_factor);
    weight_clip_factor = static_cast<T>(weight_clip_factor);

    VLOG(3) << "clip_factor=" << clip_factor << ", expand_factor=" << expand_factor;
    VLOG(3) << "weight_clip_factor=" << weight_clip_factor << ", weight_expand_factor=" << weight_expand_factor;

    cast_and_padding<T>(ctx.cuda_device_context().stream(), insnum_ori, infea_ori, insnum_pad, infea_pad, input->data<T>(), input_help.mutable_data<int8_t>(ctx.GetPlace()), scale, expand_factor, clip_factor, int8_range);
    cast_and_padding<T>(ctx.cuda_device_context().stream(), infea_ori, outfea_ori, infea_pad, outfea_pad, w->data<T>(), w_help.mutable_data<int8_t>(ctx.GetPlace()), scale, weight_expand_factor, weight_clip_factor, int8_range);
    //cast_and_padding<T>(ctx.cuda_device_context().stream(), outfea_ori, 1, outfea_pad, 1, bias->data<T>(), bias_help.mutable_data<int8_t>(ctx.GetPlace()), scale, expand_factor, clip);

    VLOG(3) << "input dim0=" << input->dims()[0] << ", input dim1=" << input->dims()[1]
            << ", input_help dim0=" << input_help.dims()[0] << ", input_help dim1=" << input_help.dims()[1];
    VLOG(3) << "w dim0=" << w->dims()[0] << ", w dim1=" << w->dims()[1]
            << ", w_help dim0=" << w_help.dims()[0] << ", w_help dim1=" << w_help.dims()[1];
    VLOG(3) << "bias dim0=" << bias->dims()[0] << ", bias dim1=" << bias->dims()[1]
            << ", bias_help dim0=" << bias_help.dims()[0] << ", bias_help dim1=" << bias_help.dims()[1];

    // end cast and pad

    CBLAS_TRANSPOSE transA = CblasNoTrans;
    CBLAS_TRANSPOSE transB = CblasNoTrans;

    auto blas = phi::funcs::GetBlas<GPUCtx, int8_t>(dev_ctx);

    T bias_scale_factor_use = static_cast<T>(1.0);
    float alpha = static_cast<float>(1);
    float beta = static_cast<float>(0);

    //blas.GEMM(transA, transB, insnum_pad, outfea_pad, infea_pad, alpha, input_help.data<paddle::platform::float16>(), w_help.data<paddle::platform::float16>(), beta, output_help.mutable_data<paddle::platform::float16>(ctx.GetPlace()));
    int flag = 0;
    blas.GEMM(transA, transB, insnum_pad, outfea_pad, infea_pad, alpha, input_help.data<int8_t>(), w_help.data<int8_t>(), beta, output_help.mutable_data<float>(ctx.GetPlace()), flag);

    T scale_factor = static_cast<T>(1.0);
    T interval = static_cast<T>(2*clip_factor/int8_range);
    //T interval = (static_cast<T>(2*clip_factor/int8_range)) * (static_cast<T>(2*weight_clip_factor/int8_range)) ;
    VLOG(3) << "interval=" << interval;
    cast_and_cut<T>(ctx.cuda_device_context().stream(), insnum_ori, outfea_ori, insnum_pad, outfea_pad, output->data<T>(), output_help.data<float>(), scale_factor, expand_factor, weight_expand_factor, interval);

    //add bias
    vec_mat_row_add<T>(ctx.cuda_device_context().stream(), insnum_ori, outfea_ori,
                       output->data<T>(), bias->data<T>(), bias_scale_factor_use);

  }
};

template <typename DeviceContext, typename T>
class ScaledINT8FCGradOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto* input = ctx.Input<Tensor>("Input");
    auto* w = ctx.Input<Tensor>("W");
    auto* dout = ctx.Input<Tensor>(framework::GradVarName("Out")); // insnum * outfea

    T bias_scale_factor_use = static_cast<T>(1.0);
    T alpha = static_cast<T>(1.0);
    T beta = static_cast<T>(0.0);

    auto* dx = ctx.Output<Tensor>(framework::GradVarName("Input"));
    auto* dw = ctx.Output<Tensor>(framework::GradVarName("W"));
    auto* db = ctx.Output<Tensor>(framework::GradVarName("Bias"));

    auto input_dims = input->dims(); //ins_num*in_feat
    auto dout_dims = dout->dims(); //ins_num*out_feat
    auto w_dims = w->dims(); //in_feat*out_feat

    auto& dev_ctx = ctx.template device_context<GPUCtx>();
    auto stream = ctx.cuda_device_context().stream();

    //init
    dx->mutable_data<T>(ctx.GetPlace());
    phi::funcs::set_constant(dev_ctx, dx, 0.0);

    dw->mutable_data<T>(ctx.GetPlace());
    phi::funcs::set_constant(dev_ctx, dw, 0.0);

    db->mutable_data<T>(ctx.GetPlace());
    phi::funcs::set_constant(dev_ctx, db, 0.0);

    // get bias grad
    auto dout_coln = dout_dims[1];
    auto ins_num = dout_dims[0];
    col_sum_mat(stream, ins_num, dout_coln, dout->data<T>(), db->data<T>(), bias_scale_factor_use);
    // dx dw
    const unsigned int insnum_ori = input_dims[0];
    const unsigned int infea_ori = input_dims[1];
    const unsigned int outfea_ori = w_dims[1];
    
    auto blas = phi::funcs::GetBlas<GPUCtx, T>(dev_ctx);
    //dx = dy * w^T
    blas.GEMM(CblasNoTrans, CblasTrans, insnum_ori, infea_ori, outfea_ori, alpha, dout->data<T>(), w->data<T>(), beta, dx->mutable_data<T>(ctx.GetPlace()));
    //dw = x^T * dy
    blas.GEMM(CblasTrans, CblasNoTrans, infea_ori, outfea_ori, insnum_ori, alpha, input->data<T>(), dout->data<T>(), beta, dw->mutable_data<T>(ctx.GetPlace()));
  }
};
}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(scaled_int8fc, ops::ScaledINT8FCCUDAKernel<GPUCtx, float>,
                        ops::ScaledINT8FCCUDAKernel<GPUCtx, double>);

REGISTER_OP_CUDA_KERNEL(scaled_int8fc_grad,
                        ops::ScaledINT8FCGradOpCUDAKernel<GPUCtx, float>,
                        ops::ScaledINT8FCGradOpCUDAKernel<GPUCtx, double>);                      
